/*
#v1
Ideia: Transformar as matrizes em transpostas para nao precisar fazer ler dois ponteiros, apenas usar o deslocamento
Resultado: Aumento de performance. Tempo 1/8 vezes o anterior #8.2 -> 1.1

#v2
Ideia: Transformar matriz em vetor para preparar para CUDA
Resultado: Perda de desempenho. Tempo 2.4 vezes o anterior #1.1 -> 2.4

#v2.1
Ideia: otimizar o codigo antes do CUDA procurando por calculos repetidos e os atribuindo a auxiliares
Resultado: Ganho de desempenho. Tempo 10/15 vezes o anterior #2.4 -> 1.55

#v3
Ideia: Utilizar a GPU para fazer os calculos quando a matriz for grande
Resultado: Ganho de desempenho. Tempo 10/50 vezes o anterior #1.55 -> 0.35

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NTRANS 0
#define TRANS 1

#define OPMAX 1024
#define THREADSMAX 32	//32*32=1024, Nao se deve ter mais que 1024 threads por bloco

struct _matriz
{
	int n;
	int m;
	int *cont;
}; typedef struct _matriz Matriz;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int*) malloc(n * m * sizeof(int*));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i * m + j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i * mat->m + j]);
		printf("\n");
	}
}

void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	int aux;
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->n; j++)
		{
			aux = i * c->m + j;
			c->cont[aux] = 0;
			for(int k = 0; k < b->m; k++)
				c->cont[aux] += a->cont[i * a->m + k] * b->cont[j * b->m + k];
		}
}

//KERNEL QUE EXECUTA AS INSTRUCOES NA GPU
__global__ void kernelMulMatriz(int *a, int nA, int mA, int *b, int nB, int mB, int *c, int mC)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if(i < nA && j < nB)
	{
		int aux = i * mC + j;
		c[aux] = 0;
		for(int k = 0; k < mB; k++)
			c[aux] += a[i * mA + k] * b[j * mB + k];
	}
}

//FUNCAO QUE PREPARA PARA A GPU
void multiplicarMatrizesCUDA(Matriz *a, Matriz *b, Matriz *c)
{
	int *d_a, *d_b, *d_c, opCount = a->n * b->n;
	dim3 blocksPerGrid(1, 1), threadsPerBlock(a->n, b->n);
	if(opCount > OPMAX)
	{
		threadsPerBlock.x = THREADSMAX;
		threadsPerBlock.y = THREADSMAX;
		blocksPerGrid.x = ceil(((double) a->n / THREADSMAX));
		blocksPerGrid.y = ceil(((double) b->n / THREADSMAX));
	}

	hipMalloc(&d_a, sizeof(int) * a->n * a->m);
	hipMalloc(&d_b, sizeof(int) * b->n * b->m);
	hipMalloc(&d_c, sizeof(int) * c->n * c->m);
	hipMemcpy(d_a, a->cont, sizeof(int) * a->n * a->m, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b->cont, sizeof(int) * b->n * b->m, hipMemcpyHostToDevice);

	kernelMulMatriz<<<blocksPerGrid, threadsPerBlock>>> (d_a, a->n, a->m, d_b, b->n, b->m, d_c, c->m);

	hipMemcpy(c->cont, d_c, sizeof(int) * c->n * c->m, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipDeviceSynchronize();
}

Matriz *lerMatriz(char *nome, int n, int m, short int trans)
{
	Matriz *mat = NULL;
	FILE *f = fopen(nome, "r");
	if(trans)
	{
		mat = criarMatriz(m, n);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[j * n + i]));
	}
	else
	{
		mat = criarMatriz(n, m);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[i * m + j]));
	}
	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat, short int trans)
{
	static int i = 0;
	char nome[100];

	if(trans)	sprintf(nome, "%d-%dx%d.txt", i, mat->m, mat->n);
	else		sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	if(trans)
		for(int i = 0; i < mat->m; i++)
		{
			for(int j = 0; j < mat->n; j++)
				fprintf(f, "%d ", mat->cont[j * mat->m + i]);
			fprintf(f, "\n");
		}
	else
		for(int i = 0; i < mat->n; i++)
		{
			for(int j = 0; j < mat->m; j++)
				fprintf(f, "%d ", mat->cont[i * mat->m + j]);
			fprintf(f, "\n");
		}

	fclose(f);
	i++;
}

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(m2, n2); //INVERTIDOS PARA A TRANSPOSTA
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1, NTRANS);
					b = lerMatriz(argv[7], n2, m2, TRANS);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoExecMul(Input *i, void mul(Matriz*, Matriz*, Matriz*))
{
	clock_t tempo = clock();
	mul(i->a, i->b, i->c);
	tempo = clock() - tempo;

	return ((double) tempo / CLOCKS_PER_SEC);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a, NTRANS);
		salvarMatriz(i->b, TRANS);
	}
	salvarMatriz(i->c, NTRANS);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

int main(int argc, char ** argv)
{
	clock_t tempo = clock();
	Input *i = lerInput(argc, argv);
	printf("Tempo de criacao: %lf\n", (((double) clock() - tempo) / CLOCKS_PER_SEC));
	printf("Tempo de execucao: %lf\n", medirTempoExecMul(i, &multiplicarMatrizesCUDA));
	salvarELiberarMatrizes(i);

	return 0;
}
